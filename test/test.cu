#include "hip/hip_runtime.h"

#include <cstdio>
#include <hip/hip_vector_types.h>


texture<float4,  1, hipReadModeElementType> texture_float_1D;
texture<unsigned char, 3, hipReadModeElementType> tex;

__device__unsigned char mul(unsigned char variable)
{
 
    return variable - 1;
}

extern "C" {
__global__ void helloWorld(unsigned char *data3, unsigned int width) {
	    
	    /*matrix mul sample*/
	    /*
		int tid, tx, ty;
		tx = blockDim.x*blockIdx.x + threadIdx.x;
		ty = blockDim.y*blockIdx.y + threadIdx.y;
		tid = width*ty + tx;
		
		if ((tx >= width) || (tx >= width)) return;
		
		int Value = 0;
	
		
		for (int i = 0; i < width; i++)
		{
		  int MVal=data1[ty * width + i];
		  int NVal=data2[i * width + tx];
		   Value += MVal * NVal;
		 
		}
		
		data3[tid]= Value;
		*/
		
		 /*1D OTF Table sample*/
		/*
		for(float i = 0; i<=1.0f; i+=1.0f/256.0f){
			float4 result = tex1D(texture_float_1D,0.31640625);
			data3[0] = result;
		
		}
		*/
		
		
		/*3D array sample*/
		/*
		int loop;
		unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
		unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
		unsigned int z;
	  
		for(loop=0; loop<2; loop++){
			z = loop;
			data3[z*2*2 + y*2 + x] = tex3D(tex, x, y, z);
		}
		*/
		
		/*3D volume sample*/
		
		int loop;
		unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
		unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
		unsigned int z;
	  
		for(loop=0; loop<225; loop++){
			z = loop;
			unsigned char result = tex3D(tex, x, y, z);
			data3[z*256*256 + y*256 + x]=mul(result); 
		}
		
		
	}
}
