#include "hip/hip_runtime.h"

#include <cstdio>
#include <hip/hip_vector_types.h>

typedef unsigned int  uint;
typedef unsigned char uchar;
typedef unsigned char VolumeType;

texture<VolumeType, 3, hipReadModeNormalizedFloat> tex;    
texture<float4,  1, hipReadModeElementType> texture_float_1D;


struct Ray
{
    float3 o;   // origin
    float3 d;   // direction
};

__device__
int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{

    float3 invR = make_float3(1.0f) / r.d;
    float3 tbot = invR * (boxmin - r.o);
    float3 ttop = invR * (boxmax - r.o);

    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

    *tnear = largest_tmin;
    *tfar = smallest_tmax;

    return smallest_tmax > largest_tmin;
}


__device__
float3 mul(const float *M, const float3 &v)
{
   float3 r;
   
   r.x = v.x * M[0] + v.y * M[1] + v.z * M[2];
   r.y = v.x * M[4] + v.y * M[5] + v.z * M[6];
   r.z = v.x * M[8] + v.y * M[9] + v.z * M[10];
   
   return r;
}

__device__
float4 mul(const float *M, const float4 &v)
{
	float4 r;

	r.x = v.x * M[0] + v.y * M[1] + v.z * M[2]  + v.w * M[3];
	r.y = v.x * M[4] + v.y * M[5] + v.z * M[6]  + v.w * M[7];
	r.z = v.x * M[8] + v.y * M[9] + v.z * M[10] + v.w * M[11];	
	r.w = 1.0f;
	
	return r;
}

__device__ uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}


extern "C" {
__global__ void render_kernel_volume(uint *d_output, 
								  float *d_invViewMatrix, 
								  unsigned int imageW,
								  unsigned int imageH,
								  float density,
								  float brightness,
								  float transferOffset,
								  float transferScale) 
{
	
		const int maxSteps = 500;
		const float tstep = 0.01f;
		const float opacityThreshold = 0.95f;
		const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
		const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);
	 
		unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
		unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	 
		if ((x >= imageW) || (y >= imageH)) return;
	 
		float u = (x / (float) imageW)*2.0f-1.0f;
		float v = (y / (float) imageH)*2.0f-1.0f;
	 
		Ray eyeRay;
		eyeRay.o = make_float3(mul(d_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
		eyeRay.d = normalize(make_float3(u, v, -2.0f));
		eyeRay.d = mul(d_invViewMatrix, eyeRay.d);
	 
		float tnear, tfar;
		int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);
	 
		if (!hit) return;
	 
		if (tnear < 0.0f) tnear = 0.0f; 
	 
		float4 sum = make_float4(0.0f);
		float t = tnear;
		float3 pos = eyeRay.o + eyeRay.d * tnear;
		float3 step = eyeRay.d*tstep;
	 
		for (float i=0; i<maxSteps; i++){
				
				float sample = tex3D(tex,pos.x*0.5f+0.5f, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);
				
				float4 col = tex1D(texture_float_1D, (sample-transferOffset)*transferScale);
     
				col.x *= col.w;
				col.y *= col.w;
				col.z *= col.w;
				
				sum = sum + col*(1.0f - sum.w);
     
				if (sum.w > opacityThreshold)
					break;
					
				t += (tstep*0.5);

				if (t > tfar) break;

				pos += (step*0.5);
  
		}
		sum.w=0.0;
		d_output[y*imageW + x] = rgbaFloatToInt(sum);
	}
}
extern "C" {
__global__ void render_kernel_MIP(uint *d_output, 
								  float *d_invViewMatrix, 
								  unsigned int imageW,
								  unsigned int imageH,
								  float density,
								  float brightness,
								  float transferOffset,
								  float transferScale) 
{
	
		const int maxSteps = 500;
		const float tstep = 0.01f;
		const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
		const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);
	 
		unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
		unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	 
		if ((x >= imageW) || (y >= imageH)) return;
	 
		float u = (x / (float) imageW)*2.0f-1.0f;
		float v = (y / (float) imageH)*2.0f-1.0f;
	 
		Ray eyeRay;
		eyeRay.o = make_float3(mul(d_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
		eyeRay.d = normalize(make_float3(u, v, -2.0f));
		eyeRay.d = mul(d_invViewMatrix, eyeRay.d);
	 
		float tnear, tfar;
		int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);
	 
		if (!hit) return;
	 
		if (tnear < 0.0f) tnear = 0.0f; 
	 
		float4 sum = make_float4(0.0f);
		float t = tnear;
		float3 pos = eyeRay.o + eyeRay.d * tnear;
		float3 step = eyeRay.d*tstep;
		float max = 0.0f; 
		for (float i=0; i<maxSteps; i++){
				
				float sample = tex3D(tex, pos.x*0.5f+0.5f, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);
				if(sample >= max) 
					max = sample;
					
				t += (tstep*0.5);

			   if (t > tfar) break;

				pos += (step*0.5);
			
		}
		sum.x = max;
		sum.y = max;
		sum.z = max;
		sum.w = 0;
		d_output[y*imageW + x] = rgbaFloatToInt(sum);
	}
}
extern "C" {
__global__ void render_kernel_MRI(uint *d_output, 
								  float *d_invViewMatrix, 
								  unsigned int imageW,
								  unsigned int imageH,
								  float density,
								  float brightness,
								  float transferOffset,
								  float transferScale) 
	{
		const int maxSteps = 500;
		const float tstep = 0.01f;
		const float opacityThreshold = 0.95f;
		const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
		const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

		uint x = blockIdx.x*blockDim.x + threadIdx.x;
		uint y = blockIdx.y*blockDim.y + threadIdx.y;

		if ((x >= imageW) || (y >= imageH)) return;

		float u = (x / (float) imageW)*2.0f-1.0f;
		float v = (y / (float) imageH)*2.0f-1.0f;

		// calculate eye ray in world space
		Ray eyeRay;
		eyeRay.o = make_float3(mul(d_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
		eyeRay.d = normalize(make_float3(u, v, -2.0f));
		eyeRay.d = mul(d_invViewMatrix, eyeRay.d);

		// find intersection with box
		float tnear, tfar;
		int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

		if (!hit) return;

		if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

		// march along ray from front to back, accumulating color
		float4 sum = make_float4(0.0f);
		float t = tnear;
		float3 pos = eyeRay.o + eyeRay.d * tnear;
		float3 step = eyeRay.d*tstep;
		
		float max = 0.0f; 
		
				
		float sample = tex3D(tex, pos.x+0.5f, pos.y+0.5f+transferOffset, pos.z+0.5f);
				
		sum.x = sample;
		sum.y = sample;
		sum.z = sample;
		sum.w = 0;
		d_output[y*imageW + x] = rgbaFloatToInt(sum);
	}
}
