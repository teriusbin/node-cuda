#include "hip/hip_runtime.h"

#include <cstdio>
#include <hip/hip_vector_types.h>

texture<float4,  1, hipReadModeElementType> texture_float_1D;
texture<unsigned char, 3, hipReadModeElementType> tex;
__device__
unsigned char mul(unsigned char variable)
{
  
   
    return variable + 1;
}

extern "C" {
__global__ void render_kernel(float *data3, 
								  float *d_invViewMatrix, 
								  unsigned int imageW,
								  unsigned int imageH,
								  float density,
						         float brightness,
						         float transferOffset,
						         float transferScale) {
	    
	    /*matrix mul sample*/
	    /*
		int tid, tx, ty;
		tx = blockDim.x*blockIdx.x + threadIdx.x;
		ty = blockDim.y*blockIdx.y + threadIdx.y;
		tid = width*ty + tx;
		
		if ((tx >= width) || (tx >= width)) return;
		
		int Value = 0;
	
		
		for (int i = 0; i < width; i++)
		{
		  int MVal=data1[ty * width + i];
		  int NVal=data2[i * width + tx];
		   Value += MVal * NVal;
		 
		}
		
		data3[tid]= Value;
		*/
		
		 /*1D OTF Table sample*/
		/*
		for(float i = 0; i<=1.0f; i+=1.0f/256.0f){
			float4 result = tex1D(texture_float_1D,0.31640625);
			data3[0] = result;
		
		}
		*/
		
		/*3D array sample*/
		/*
		int loop;
		unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
		unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
		unsigned int z;
	  
		for(loop=0; loop<2; loop++){
			z = loop;
			data3[z*2*2 + y*2 + x] = tex3D(tex, x, y, z);
		}
		*/
		
		
		/*3D volume sample*/
		/*
		int loop;
		unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
		unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
		unsigned int z;
	  
		for(loop=0; loop<225; loop++){
			z = loop;
			unsigned char result = tex3D(tex, x, y, z);
			data3[z*256*256 + y*256 + x]=mul(result); 
		}
		*/
		
		/*d_invViewMatrix*/
		/*
		for (int i=0; i<12; i++) {
			data3[i] = d_invViewMatrix[i];
		}
		*/
		
		/*variable value*/
		data3[0] = density;
		data3[1] = brightness;
		data3[2] = transferOffset;
		data3[3] = transferScale;
		
		
		
	
	}
}
